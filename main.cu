#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
    	// Define blocks and threads then calculate
    	int i = blockIdx.x * blockDim.x + threadIdx.x;
    	
	for(int j = 0; j < 100000; j++)
		if (i < n) y[i] = a*x[i] + y[i];

    	__syncthreads();
}

int main(void)
{
    // N = 1M
    int N = 1<<23;

    // Create host and device memory
    float *x, *y, *d_x, *d_y;
    x = (float*)malloc(N*sizeof(float));
    y = (float*)malloc(N*sizeof(float));

    hipMalloc(&d_x, N*sizeof(float)); 
    hipMalloc(&d_y, N*sizeof(float));

    // Assign local memory
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Copy memory to device
    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

    // Perform SAXPY on 1M elements
    saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

    // Copy completed arithmetic back to host
    hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

    // Clean up device
    hipDeviceReset();

    // Check for errors
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = max(maxError, abs(y[i]-200002.0f));

    printf("Max error: %f\n", maxError);
    printf("%f\n", y[1]);

    // Free memory
    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);

    return 0;
}
